#include "hip/hip_runtime.h"
/* 
Simplest possible Cuda-like Cuda program.
Adds two GPU vectors into one new GPU vector. 
"vectorAdd" function is the only part that needs to be modified to execute any 
function that takes two input vectors and returns one output vector.
For clarity, no error checking is used.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements) {
		C[i] = A[i] + B[i];
	}
}

int main(void)
{
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);
	float *h_A = (float *)malloc(size);     // Allocate host input vector A
	float *h_B = (float *)malloc(size);     // Allocate host input vector B
	float *h_C = (float *)malloc(size);     // Allocate host output vector C
	for (int i = 0; i < numElements; ++i) /* Initialize the host input vectors*/ {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	printf("%f\n", h_A[1]);
	printf("%f\n", h_B[1]);

	float *d_A = NULL; // Allocate the device input vector A
	hipMalloc((void **)&d_A, size);
	float *d_B = NULL;// Allocate the device input vector B
	hipMalloc((void **)&d_B, size);
	float *d_C = NULL; // Allocate the device output vector C
	hipMalloc((void **)&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	vectorAdd <<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements); 
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("%f\n", h_C[1]);

	hipFree(d_A); // Free device global memory
	hipFree(d_B);
	hipFree(d_C);
	free(h_A); // Free host memory
	free(h_B);
	free(h_C);
	printf("Done\n");
	return 0;
}

